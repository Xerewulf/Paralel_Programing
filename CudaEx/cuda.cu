
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

// CUDA kernel. Each thread takes care of one element of c
__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

void Serial_sum(double *a, double *b, double cz[], int n)
{

for(int i=0;i<n;i++){
cz[i]= a[i]+b[i];
 }

}

double twoNormDiff(double *c,double *cz, int n)
{
double sum=0;
for(int i=0;i<n;i++){
	sum += pow(c[i],2)-pow(cz[i],2);
    }
return sqrt(sum);
}





int main( int argc, char* argv[] )
{
    // Size of vectors
    int n = 500000;
 
    // Host input vectors
    double *h_a;
    double *h_b;
    //Host output vector
    double *h_c;
    double *cz;
    // Device input vectors
    double *d_a;
    double *d_b;
    //Device output vector
    double *d_c;

    // Size, in bytes, of each vector
    size_t bytes = n*sizeof(double);

    // Allocate memory for each vector on host
    h_a = (double*)malloc(bytes);
    h_b = (double*)malloc(bytes);
    h_c = (double*)malloc(bytes);
    cz  = (double*)malloc(bytes);
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);

    int i;
    // Initialize vectors on host
    for( i = 0; i < n; i++ ) {
        h_a[i] = (rand()%50000)*sin(i);
        h_b[i] = (rand()%50000)*cos(i);
    }
 
    // Copy host vectors to device
    hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
    int blockSize, gridSize;
 
    // Number of threads in each thread block
    blockSize = strtol(argv[1], NULL, 10);
    gridSize  = strtol(argv[2], NULL, 10);

    // Number of thread blocks in grid
    
 
    // Execute the kernel
    vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
    // Copy array back to host
    hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
    Serial_sum(d_a, d_b, cz, n);
    printf("two norm of diffence between howstt and device is: %lf", twoNormDiff(h_c, cz, n));
 
    // Release device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
 
    // Release host memory
    free(h_a);
    free(h_b);
    free(h_c);
 
    return 0;
}
